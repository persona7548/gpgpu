#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define LEN_M 4
#define LEN_N 4
#define LEN_K 3

#define TILE_WIDTH 2

__global__ void MatrixMulOnDevice(int m, int n, int k, float * A, float * B, float * C) {
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if ((row<m) && (col<k)) {
		float temp = 0.0;
		for (int i = 0; i<n; i++) {
			temp += A[row*n + i] * B[i*k + col];
		}
		C[row*k + col] = temp;
	}
}

int main()
{
	// Allocate and initialize the matrices A, B, C
	float * A, *B, *C;

	A = (float*)malloc(LEN_M*LEN_N * sizeof(float));
	B = (float*)malloc(LEN_N*LEN_K * sizeof(float));
	C = (float*)malloc(LEN_M*LEN_K * sizeof(float));

	for (int i = 0; i<LEN_M*LEN_N; i++) A[i] = i;
	for (int i = 0; i<LEN_N*LEN_K; i++) B[i] = i;
	for (int i = 0; i<LEN_M*LEN_K; i++) C[i] = 0.0;

	// I/O to read the input matrices A and B
	float * dev_A, *dev_B, *dev_C;
	hipMalloc((void**)&dev_A, LEN_M*LEN_N * sizeof(float));
	hipMalloc((void**)&dev_B, LEN_N*LEN_K * sizeof(float));
	hipMalloc((void**)&dev_C, LEN_M*LEN_K * sizeof(float));


	hipMemcpy(dev_A, A, LEN_M*LEN_N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, B, LEN_N*LEN_K * sizeof(float), hipMemcpyHostToDevice);

	// A*B on the device
	dim3 dimGrid((LEN_K - 1) / TILE_WIDTH + 1, (LEN_M - 1) / TILE_WIDTH + 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	MatrixMulOnDevice << <dimGrid, dimBlock >> >(LEN_M, LEN_N, LEN_K, dev_A, dev_B, dev_C);

	// I/O to write the output matrix C
	hipMemcpy(C, dev_C, LEN_M*LEN_K * sizeof(float), hipMemcpyDeviceToHost);

	printf("C:");
	for (int i = 0; i<LEN_M*LEN_K; i++) printf(" %6.1f", C[i]);
	printf("\n");

	// Free matrices A, B, C
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);

	free(A);
	free(B);
	free(C);
	return 0;
}

